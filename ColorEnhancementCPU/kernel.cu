#include "hip/hip_runtime.h"
/*
* Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

// This example implements the contrast adjustment on an 8u one-channel image by using
// Nvidia Performance Primitives (NPP). 
// Assume pSrc(i,j) is the pixel value of the input image, nMin and nMax are the minimal and 
// maximal values of the input image. The adjusted image pDst(i,j) is computed via the formula:
// pDst(i,j) = (pSrc(i,j) - nMin) / (nMax - nMin) * 255 
//
// The code flow includes five steps:
// 1) Load the input image into the host array;
// 2) Allocate the memory space on the GPU and copy data from the host to GPU;
// 3) Call NPP functions to adjust the contrast;
// 4) Read data back from GPU to the host;
// 5) Output the result image and clean up the memory.

#include <iostream>
#include <fstream>
#include <sstream>
#include "npp.h"
#include <windows.h>

double PCFreq = 0.0;
__int64 CounterStart = 0;

void StartCounter()
{
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		std::cout << "QueryPerformanceFrequency failed!\n";

	PCFreq = double(li.QuadPart) / 1000000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter()
{
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}

struct MinMax {
	Npp8u min;
	Npp8u max;
};
// Function declarations.
Npp8u *
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray);

void
WritePGM(char * sFileName, Npp8u * pDst_Host, int nWidth, int nHeight, int nMaxGray);

MinMax 
MinMaxCalc( Npp8u * pSrc_Host, int & nWidth, int & nHeight);

void
SubtractMin(Npp8u * pDst_Host, Npp8u  nMin, Npp8u * pSrc_Host, int & nWidth, int & nHeight);

void 
MultiplyConstantDivideScaleFactor(Npp8u * pDst_Host, Npp8u  nConstant, int nScaleFactor, int & nWidth, int & nHeight);

// Main function.
int
main(int argc, char ** argv)
{
	// Host parameter declarations.	
	Npp8u * pSrc_Host, *pDst_Host;
	int   nWidth, nHeight, nMaxGray;

	// Load image to the host.
	std::cout << "Load PGM file." << std::endl;
	pSrc_Host = LoadPGM((char *)"C:\\Users\\blgnksy\\source\\repos\\CudaAssignment2\\ColorEnhancement\\lena_before.pgm", nWidth, nHeight, nMaxGray);
	pDst_Host = new Npp8u[nWidth * nHeight];

	std::cout << "Process the image on CPU." << std::endl;
	StartCounter();

	/*CPU Min Max Calculator*/
	MinMax mm=MinMaxCalc(pSrc_Host, nWidth, nHeight);
	printf("%d\t%d\n", mm.min, mm.max);
	int nScaleFactor = 0;
	int nPower = 1;
	while (nPower * 255.0f / (mm.max - mm.min) < 255.0f)
	{
		nScaleFactor++;
		nPower *= 2;
	}

	Npp8u nConstant = static_cast<Npp8u>(255.0f / (mm.max - mm.min) * (nPower / 2));
	printf("Constant is %d.\n", nConstant);

	SubtractMin(pDst_Host, mm.min, pSrc_Host, nWidth, nHeight);

	MultiplyConstantDivideScaleFactor(pDst_Host, nConstant, nScaleFactor,  nWidth, nHeight);

	std::cout<<GetCounter()<<" ms."<<std::endl;
	std::cout << "Work done!" << std::endl;

	// Output the result image.
	std::cout << "Output the PGM file." << std::endl;
	WritePGM((char *)"C:\\Users\\blgnksy\\source\\repos\\CudaAssignment2\\lena_afterCPU3.pgm", pDst_Host, nWidth, nHeight, nMaxGray);

	// Clean up.
	std::cout << "Clean up." << std::endl;
	delete[] pSrc_Host;
	delete[] pDst_Host;


	getchar();
	return 0;
}

// Disable reporting warnings on functions that were marked with deprecated.
#pragma warning( disable : 4996 )

// Load PGM file.
Npp8u *
LoadPGM(char * sFileName, int & nWidth, int & nHeight, int & nMaxGray)
{
	char aLine[256];
	FILE * fInput = fopen(sFileName, "r");
	if (fInput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	// First line: version
	fgets(aLine, 256, fInput);
	std::cout << "\tVersion: " << aLine;
	// Second line: comment
	fgets(aLine, 256, fInput);
	std::cout << "\tComment: " << aLine;
	fseek(fInput, -1, SEEK_CUR);
	// Third line: size
	fscanf(fInput, "%d", &nWidth);
	std::cout << "\tWidth: " << nWidth;
	fscanf(fInput, "%d", &nHeight);
	std::cout << " Height: " << nHeight << std::endl;
	// Fourth line: max value
	fscanf(fInput, "%d", &nMaxGray);
	std::cout << "\tMax value: " << nMaxGray << std::endl;
	while (getc(fInput) != '\n');
	// Following lines: data
	Npp8u * pSrc_Host = new Npp8u[nWidth * nHeight];
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			pSrc_Host[i*nWidth + j] = fgetc(fInput);
	fclose(fInput);

	return pSrc_Host;
}

// Write PGM image.
void
WritePGM(char * sFileName, Npp8u * pDst_Host, int nWidth, int nHeight, int nMaxGray)
{
	FILE * fOutput = fopen(sFileName, "w+");
	if (fOutput == 0)
	{
		perror("Cannot open file to read");
		exit(EXIT_FAILURE);
	}
	char * aComment = (char *)"# Created by Bilgin Aksoy CUDA Assignment";
	fprintf(fOutput, "P5\n%s\n%d %d\n%d\n", aComment, nWidth, nHeight, nMaxGray);
	for (int i = 0; i < nHeight; ++i)
		for (int j = 0; j < nWidth; ++j)
			fputc(pDst_Host[i*nWidth + j], fOutput);
	fclose(fOutput);
}

MinMax 
MinMaxCalc( Npp8u * pSrc_Host, int & nWidth, int & nHeight) {
	Npp8u min = 0;
	Npp8u max = 0;
	for (int i = 0; i < nHeight; i++)
	{
		for (int j = 0; j < nWidth; j++)
		{
			if (i ==0 && j == 0)
			{
				min = pSrc_Host[i*nWidth + j];
			}
			if (pSrc_Host[i*nWidth + j] <= min)
			{
				min = pSrc_Host[i*nWidth + j];
			}
			if (pSrc_Host[i*nWidth + j] >= max)
			{
				max = pSrc_Host[i*nWidth + j];
			}
		}
	}

	MinMax mm = { min,max };
	printf("Min Value= %d Max Value=%d for the given image.\n", mm.min, mm.max);
	//getchar();
	return mm;
}

void SubtractMin(Npp8u * pDst_Host, Npp8u   nMin, Npp8u * pSrc_Host, int & nWidth, int & nHeight) {
	for (int i = 0; i < nHeight; i++)
	{
		for (int j = 0; j < nWidth; j++)
		{
			//printf("Before subtract %d\n", pDst_Host[i*nWidth + j]);
			pDst_Host[i*nWidth + j] = pSrc_Host[i*nWidth + j] - nMin;
			//printf("After subtract %d\n", pDst_Host[i*nWidth + j]);
		}
	}
	//printf("Mininum Value Subtracted...\n");
	//getchar();
}

void MultiplyConstantDivideScaleFactor(Npp8u * pDst_Host, Npp8u  nConstant, int nScaleFactor,  int & nWidth, int & nHeight) {
	for (int i = 0; i < nHeight; i++)
	{
		for (int j = 0; j < nWidth; j++)
		{
			//printf("Before multiply %d\n", pDst_Host[i*nWidth + j]);
			pDst_Host[i*nWidth + j] = pDst_Host[i*nWidth + j] * nConstant/(nScaleFactor-1);
			//printf("After multiply %d\n", pDst_Host[i*nWidth + j]);
		}
	}
	//printf("Constant Value Multiplied...\n");
	//getchar();
}